
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 32


void cpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    for(int y=0; y<size; ++y)
    {
        for(int x=0; x<size; ++x)
        {
            int tmp = 0;
            for(int step = 0; step < size; ++step)
            {
                tmp += a[y*size + step] * b[step * size + x];
            }
            c[y * size + x] = tmp;
        }
    }
}

__global__ void gpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int tmp = 0;
    if( x < size && y < size)
    {
        for( int step = 0; step < size; ++step)
        {
            tmp += a[y * size + step] * b[step * size + x];
        }
        c[y * size + x] = tmp;
    }
}



int main()
{
    int matrix_size = 1000;
    int memsize = sizeof(int) * matrix_size * matrix_size;


    int *h_a, *h_b, *h_c, *h_cc;

    hipHostMalloc( (void**)&h_a, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&h_b, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&h_c, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&h_cc, memsize, hipHostMallocDefault);

    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            h_a[y * matrix_size + x] = rand() % 1024;
        }
    }

    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            h_b[y * matrix_size + x] = rand() % 1024;
        }
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a , memsize);
    hipMalloc((void**) &d_b , memsize);
    hipMalloc((void**) &d_c , memsize);


    hipMemcpy( d_a, h_a, memsize, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, memsize, hipMemcpyHostToDevice);


    unsigned int grid_rows = (matrix_size +BLOCK_SIZE -1)/BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size +BLOCK_SIZE -1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);//1.gpu warp 32 2. <= 1024

    gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);

    hipMemcpy( h_c, d_c, memsize, hipMemcpyDeviceToHost);


    cpu_matrix_mult(h_a, h_b, h_cc, matrix_size);






    bool errors = false;
    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            if(fabs(h_cc[y*matrix_size + x] - h_c[y*matrix_size + x]) > (1.0e-10))
            {
                //printf("%d, %d\n", y, x);
                errors = true;
            }
        }
    }
    printf("Result: %s\n", errors?"Errors":"Passed");

    hipHostFree(h_a );
    hipHostFree(h_b );
    hipHostFree(h_c );
    hipHostFree(h_cc );
    hipFree(d_a );
    hipFree(d_b );
    hipFree(d_c );
    return 0;

}


