
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>

#define BLOCK_SIZE 32
#define M 3000
#define N 1000

__managed__ int matrix[N][M];
__managed__ int gpu_result[M][N];
__managed__ int cpu_result[M][N];

__global__ void gpu_matrix_transpose(int in[N][M], int out[M][N])
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if( x < M && y < N)
    {
        out[x][y] = in[y][x];
    }
}

__global__ void gpu_shared_matrix_transpose(int in[N][M], int out[M][N])
{
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int x = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ int ken[BLOCK_SIZE+1][BLOCK_SIZE+1];//ken[32] warp

    if(x < M && y < N)
    {
        ken[threadIdx.y][threadIdx.x] = in[y][x];
    }
    __syncthreads();

    int x1 = threadIdx.x + blockDim.y * blockIdx.y;
    int y1 = threadIdx.y + blockDim.x * blockIdx.x;
    if(x1 < N && y1 < M)
    {
        out[y1][x1] = ken[threadIdx.x][threadIdx.y];//32 bank
    }

}

void cpu_matrix_transpose(int in[N][M], int out[M][N])
{
    for(int y = 0; y < N; y++)
    {
        for(int x = 0; x < M; x++)
        {
            out[x][y] = in[y][x];
        }
    }
}

int main()
{
    for(int y=0; y<N; y++)
    {
        for(int x=0; x<M; x++)
        {
            matrix[y][x] = rand()%1024;
        }
    }

    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    dim3 dimGrid((M + BLOCK_SIZE - 1)/BLOCK_SIZE, (N + BLOCK_SIZE -1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    for(int i = 0; i < 20; i++)
    {
        gpu_shared_matrix_transpose<<<dimGrid,dimBlock>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_transpose(matrix, cpu_result);

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool errors = false;
    for(int y = 0; y<M; y++)
    {
        for (int x = 0; x < N; x++)
        {
            if(fabs(cpu_result[y][x] - gpu_result[y][x]) > (1.0e-10))
            {
                errors = true;
            }
        }
        
    }

    printf("Result: %s\n", errors?"Error":"Pass");
    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20.0);

    return 0;
}







